// includes, system

#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <chrono>
#include <iomanip>
#include <cstdlib>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 3

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

using Duration = std::chrono::duration<float>;

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char * argv[]) 
{
    hipSetDevice(MYDEVICE);
    // pointer and dimension for host memory
    char* end;
    int64_t dim = std::strtoll(argv[1], &end, 10);
    int64_t dimA = dim;
    float *h_a;

    std::cout << "dim " << dimA << std::endl;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    // it has the same syntax as cudaMalloc, but it enables asynchronous copies
    size_t memSize = dimA*sizeof(float);
    //h_a = (float *) malloc(memSize);
    hipHostMalloc(&h_a, memSize, hipHostMallocDefault);
    for (int i = 0; i<dimA; ++i)
    {
        h_a[i] = i;
    }

    // Part 1 of 5: allocate device memory
    hipMalloc(&d_a, memSize);
    hipMalloc(&d_b, memSize);

    // Part 2 of 5: host to device memory copy
    auto const start1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    auto const end1 = std::chrono::high_resolution_clock::now();
    Duration d = end1-start1;

    std::cout << std::setprecision(5) << float(memSize)/(8000000000*d.count()) << "GB/s" << std::endl;

    // Part 3 of 5: device to device memory copy
    hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);

    // clear host memory
    for (int i=0; i<dimA; ++i )
    {
        h_a[i] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (int i=0; i<dimA; ++i)
    {
        assert(h_a[i] == (float) i);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree(d_a);
    hipFree(d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    //free(h_a);
    hipHostFree(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
